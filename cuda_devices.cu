#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

int main(){
	printf("Press Ctrl+z to suspend program\n");
	printf("Type bg to send program to background\n");
	printf("Attach to process with cuda-gdb\n\n");
	printf("cuda-gdb --pid=%d\n\n", getpid());
	printf("Issue the following commands in (cuda-gdb)\n");
	printf("   set var dummy=1\n");
	printf("   continue\n");
	fflush(stdout);
	volatile int dummy=0;
	int dummy2=0;
	while(!dummy){
		dummy2=1;
	}
	dummy=dummy2;

	// Initialize
	hipInit(0);

	// Get number of devices supporting CUDA
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("There is no device supporting CUDA.\n");
		exit (0);
	}
	printf("There are %d CUDA devices\n", deviceCount);
	return 0;
}
